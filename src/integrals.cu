#include "hip/hip_runtime.h"
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cassert>

#include "integrals.cuh"
#include "cuda_api.hpp"


namespace jr::calc::cuda{

template<typename Function>
__global__
void kernel(Function const& function){
	//function();
}

auto riemann_integral(
		cuda_function_wrapper const& wrapper,
		std::vector<thrust::pair<double, double>> const& ranges, 
		std::vector<double> const& deltas
) -> double{

	assert(ranges.size() == deltas.size());

	
	thrust::device_vector<thrust::pair<double, double>> ranges_dev(ranges.begin(), ranges.end());
	thrust::device_vector<double> deltas_dev(deltas.begin(), deltas.end());


	auto zipped = thrust::make_zip_iterator(thrust::make_tuple(ranges_dev.begin(), deltas_dev.begin()));



	return 0;

}

}
