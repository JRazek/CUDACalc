#include "hip/hip_runtime.h"
#include <cstdio>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cassert>
#include "cuda_api.hpp"

#include "user_kernels.hpp"

#ifdef USER_KERNEL_FILE
#endif


namespace jr::calc::cuda{

template<typename Function>
__global__
void kernel(Function const& function){
	auto id=threadIdx.x;

	std::printf("thread_id=%d\n", id);

	function();
}

auto riemann_integral(
		cuda_function_wrapper const& wrapper,
		std::vector<thrust::pair<double, double>> const& ranges, 
		std::vector<double> const& deltas
) -> double{

	assert(ranges.size() == deltas.size());

	
	thrust::device_vector<thrust::pair<double, double>> ranges_dev(ranges.begin(), ranges.end());
	thrust::device_vector<double> deltas_dev(deltas.begin(), deltas.end());


	auto zipped = thrust::make_zip_iterator(thrust::make_tuple(ranges_dev.begin(), deltas_dev.begin()));

	auto runnable = []__device__(){
		return 1;
	};


	auto samples_count=1;


	kernel<<<10,1>>>(wrapper);	

	return 0;

}

}
